
#include "ImageClass.cuh"

int CodeBaseUnified::Graphics::Image::getWidth() const {
    return width;
}

void CodeBaseUnified::Graphics::Image::setWidth(int width) {
    Image::width = width;
}

int CodeBaseUnified::Graphics::Image::getHeight() const {
    return height;
}

void CodeBaseUnified::Graphics::Image::setHeight(int height) {
    Image::height = height;
}

int *CodeBaseUnified::Graphics::Image::getImage1DArray() const {
    return image1DArray;
}

void CodeBaseUnified::Graphics::Image::setImage1DArray(int *image1DArray) {
    Image::image1DArray = image1DArray;
}

CodeBaseUnified::Graphics::imageType CodeBaseUnified::Graphics::Image::getImgType() const {
    return imgType;
}

void CodeBaseUnified::Graphics::Image::setImgType(CodeBaseUnified::Graphics::imageType imgType) {
    Image::imgType = imgType;
}
